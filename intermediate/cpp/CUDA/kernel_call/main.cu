
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

class Image {
public:
    Image(int width, int height) : width_(width), height_(height) {
        auto error = hipMalloc(&device_image_data,  width * height * sizeof(float));

        error = hipMalloc(&device_image_data,  width * height * sizeof(float));
        if(error != hipSuccess) {
             std::cout << "Error: hipMalloc failed" << std::endl;
       }
    }

    ~Image() {
        hipFree(device_image_data);
    }

private:
    float *device_image_data;
    int width_;
    int height_;
};

int main() {
    int *device_data_c = nullptr;

    auto error = hipSuccess;
    error = hipMalloc(&device_data_c, sizeof(int));

    if(error != hipSuccess) {
        std::cout << "Error: hipMalloc failed" << std::endl;
        return 1;
    }

    add<<<1,1>>>(2, 7, device_data_c);

    int c;
    error = hipMemcpy(&c, device_data_c, sizeof(int), hipMemcpyDeviceToHost);

    if(error != hipSuccess) {
        std::cout << "Error" << std::endl;
        return 1;
    }

    Image image{4096, 4096};

    std::cout << c << std::endl;
    hipFree(device_data_c);
    return 0;
}
